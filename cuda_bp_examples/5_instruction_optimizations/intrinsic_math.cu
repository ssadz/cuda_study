#include <iostream>
#include <vector>
#include <cmath>              // 用于 __sinf、rsqrtf
#include <iomanip>
#include <chrono>
#include <hip/hip_runtime.h>


// 检查 CUDA 错误的辅助宏
#define CUDA_CHECK(err) \
    do { \
        hipError_t err_ = (err); \
        if (err_ != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__; \
            std::cerr << ": " << hipGetErrorString(err_) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// 核函数：演示 CUDA 内建数学函数
__global__ void intrinsicMathKernel(float* data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float val = data[idx];
        float rsqrt_val = rsqrtf(val);
        data[idx] = __sinf(val) + val * rsqrt_val; // 内建函数
    }
}

int main() {
    const int N = 1024 * 1024 * 10;
    const size_t bytes = N * sizeof(float);

    // 初始化主机数据
    std::vector<float> h_data(N, 0.0f);
    for (int i = 0; i < N; ++i) h_data[i] = static_cast<float>(i % 100) + 0.1f;

    // 分配设备内存并拷贝
    float *d_data, *d_result;
    CUDA_CHECK(hipMalloc(&d_data, bytes));
    CUDA_CHECK(hipMalloc(&d_result, bytes));
    CUDA_CHECK(hipMemcpy(d_data, h_data.data(), bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_result, h_data.data(), bytes, hipMemcpyHostToDevice));

    // 执行配置
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // 计时
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.0f;

    CUDA_CHECK(hipEventRecord(start));
    intrinsicMathKernel<<<gridSize, blockSize>>>(d_result, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));

    std::cout << "Intrinsic Math (__sinf + val*__rsqrtf): " << ms << " ms" << std::endl;

    // 清理
    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_result));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
} 