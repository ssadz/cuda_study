#include <iostream>
#include <hip/hip_runtime.h>


// 检查 CUDA 错误的辅助宏
#define CUDA_CHECK(err) \
    do { \
        hipError_t err_ = (err); \
        if (err_ != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__; \
            std::cerr << ": " << hipGetErrorString(err_) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// 标准加载核函数：每个线程加载一个 float
__global__ void normalLoadKernel(const float* __restrict__ in, float* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx];
    }
}

// __ldg 加载核函数：每个线程使用只读缓存加载一个 float
__global__ void ldgLoadKernel(const float* __restrict__ in, float* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = __ldg(&in[idx]);
    }
}

int main() {
    const int N = 1 << 22; // 4M elements (~16 MB)
    size_t bytes = N * sizeof(float);

    // 分配页锁定主机内存并初始化
    float *h_in, *h_out;
    CUDA_CHECK(hipHostAlloc(&h_in, bytes, hipHostMallocDefault));
    CUDA_CHECK(hipHostAlloc(&h_out, bytes, hipHostMallocDefault));
    for (int i = 0; i < N; ++i) h_in[i] = static_cast<float>(i);

    // 分配设备内存
    float *d_in, *d_out;
    CUDA_CHECK(hipMalloc(&d_in, bytes));
    CUDA_CHECK(hipMalloc(&d_out, bytes));
    CUDA_CHECK(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));

    // 执行配置
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float msNormal = 0.0f, msLDG = 0.0f;

    // 普通加载测时
    CUDA_CHECK(hipEventRecord(start));
    normalLoadKernel<<<gridSize, blockSize>>>(d_in, d_out, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&msNormal, start, stop));

    // __ldg 加载测时
    CUDA_CHECK(hipEventRecord(start));
    ldgLoadKernel<<<gridSize, blockSize>>>(d_in, d_out, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&msLDG, start, stop));

    // 输出带宽对比
    std::cout << "Normal load time: " << msNormal << " ms, bandwidth: "
              << (bytes / 1e9f) / (msNormal / 1000.0f) << " GB/s" << std::endl;
    std::cout << "__ldg load time: " << msLDG << " ms, bandwidth: "
              << (bytes / 1e9f) / (msLDG / 1000.0f) << " GB/s" << std::endl;

    // 清理
    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
    CUDA_CHECK(hipHostFree(h_in));
    CUDA_CHECK(hipHostFree(h_out));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
} 