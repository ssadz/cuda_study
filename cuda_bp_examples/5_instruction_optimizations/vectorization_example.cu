#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>


// Vectorization 示例
// 教学：使用 float4 一次加载 4 个 float，减少内存事务次数，提升带宽利用率

// 标量拷贝核函数：每个线程拷贝一个 float
__global__ void scalarCopyKernel(const float* in, float* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx];
    }
}

// 向量化拷贝核函数：每个线程拷贝一个 float4
__global__ void vectorCopyKernel(const float4* in, float4* out, int n4) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n4) {
        out[idx] = in[idx];
    }
}

int main() {
    // 元素数量须为 4 的倍数
    const int N = 1 << 22; // 4M floats (~16 MB)
    const int N4 = N / 4;
    size_t bytes = N * sizeof(float);
    size_t bytes4 = N4 * sizeof(float4);

    // 分配页锁定主机内存
    float* h_in;
    float* h_out;
    hipHostAlloc(&h_in, bytes, hipHostMallocDefault);
    hipHostAlloc(&h_out, bytes, hipHostMallocDefault);
    for (int i = 0; i < N; ++i) h_in[i] = static_cast<float>(i);

    // 分配设备内存
    float *d_in_scalar, *d_out_scalar;
    float4 *d_in_vec, *d_out_vec;
    hipMalloc(&d_in_scalar, bytes);
    hipMalloc(&d_out_scalar, bytes);
    hipMalloc(&d_in_vec, bytes4);
    hipMalloc(&d_out_vec, bytes4);

    // 拷贝输入数据到设备
    hipMemcpy(d_in_scalar, h_in, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_in_vec, h_in, bytes, hipMemcpyHostToDevice); // reinterpret as float4*

    // 执行配置
    int blockSize = 256;
    int gridScalar = (N + blockSize - 1) / blockSize;
    int gridVec    = (N4 + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float msScalar = 0.0f, msVec = 0.0f;

    // 标量拷贝测时
    hipEventRecord(start);
    scalarCopyKernel<<<gridScalar, blockSize>>>(d_in_scalar, d_out_scalar, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msScalar, start, stop);

    // 向量化拷贝测时
    hipEventRecord(start);
    vectorCopyKernel<<<gridVec, blockSize>>>(d_in_vec, d_out_vec, N4);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msVec, start, stop);

    // 输出结果
    std::cout << "Scalar copy time: " << msScalar << " ms, bandwidth: "
              << (bytes / 1e9f) / (msScalar / 1000.0f) << " GB/s" << std::endl;
    std::cout << "Vector copy time: " << msVec << " ms, bandwidth: "
              << (bytes / 1e9f) / (msVec / 1000.0f) << " GB/s" << std::endl;

    // 清理
    hipFree(d_in_scalar);
    hipFree(d_out_scalar);
    hipFree(d_in_vec);
    hipFree(d_out_vec);
    hipHostFree(h_in);
    hipHostFree(h_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
} 